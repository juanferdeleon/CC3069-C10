/*
 ============================================================================
 Author        : G. Barlas
 Version       : 1.0
 Last modified : December 2014
 License       : Released under the GNU GPL 3.0
 Description   :
 To build use  : nvcc vectorAdd.cu -o vectorAdd
 ============================================================================
 */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

static const int BLOCK_SIZE = 256;
static const int N = 268435456;

#define CUDA_CHECK_RETURN(value) {           \
    hipError_t _m_cudaStat = value;         \
    if (_m_cudaStat != hipSuccess) {        \
         fprintf(stderr, "Error %s at line %d in file %s\n",              \
                 hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);    \
         exit(1);                                                         \
       } }

__global__ void vadd (int *a, int *b, int *c, int N)
{
  int myID = blockIdx.x * blockDim.x + threadIdx.x;
  // PARA INCISO 1, DESCOMENTAR LAS SIGUIENTES 4 LÍNEAS
  extern __shared__ int s_in1[];
  s_in1[threadIdx.x] = a[myID];
  s_in1[threadIdx.x+BLOCK_SIZE] = b[myID];
  
  if (myID < N) {
    c[myID] = s_in1[threadIdx.x] + s_in1[threadIdx.x+BLOCK_SIZE];
  }
}

int main (void)
{
  int *ha, *hb, *hc, *da, *db, *dc;     // host (h*) and device (d*) pointers
  int i;
  struct timeval t1, t2;
  ha = new int[N];
  hb = new int[N];
  hc = new int[N];

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  CUDA_CHECK_RETURN (hipMalloc ((void **) &da, sizeof (int) * N)); 
  CUDA_CHECK_RETURN (hipMalloc ((void **) &db, sizeof (int) * N));
  CUDA_CHECK_RETURN (hipMalloc ((void **) &dc, sizeof (int) * N));

  for (i = 0; i < N; i++)
    {
      ha[i] = rand () % 10000;
      hb[i] = rand () % 10000;
    }

  CUDA_CHECK_RETURN (hipMemcpy (da, ha, sizeof (int) * N, hipMemcpyHostToDevice));
  CUDA_CHECK_RETURN (hipMemcpy (db, hb, sizeof (int) * N, hipMemcpyHostToDevice));

  int grid = ceil (N * 1.0 / BLOCK_SIZE);
  gettimeofday(&t1, 0);
  vadd <<< grid, BLOCK_SIZE, BLOCK_SIZE*sizeof(int)*2>>> (da, db, dc, N);
  gettimeofday(&t2, 0);
  double time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0;

  printf("Tiempo que se tardo en generar:  %e ms \n", time);

  CUDA_CHECK_RETURN (hipDeviceSynchronize ());
  CUDA_CHECK_RETURN (hipGetLastError ());
  CUDA_CHECK_RETURN (hipMemcpy (hc, dc, sizeof (int) * N, hipMemcpyDeviceToHost));

  for (i = 0; i < N; i++)
    {
      if (hc[i] != ha[i] + hb[i])
        printf ("Error at index %i : %i VS %i\n", i, hc[i], ha[i] + hb[i]);
    }

  CUDA_CHECK_RETURN (hipFree ((void *) da));
  CUDA_CHECK_RETURN (hipFree ((void *) db));
  CUDA_CHECK_RETURN (hipFree ((void *) dc));
  delete[]ha;
  delete[]hb;
  delete[]hc;
  CUDA_CHECK_RETURN (hipDeviceReset ());

  return 0;
}
