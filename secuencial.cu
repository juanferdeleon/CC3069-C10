#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

static const int N = 268435456;

//hope this workss

int main (void)
{
    int *ha, *hb, *hc; 
    int i;
    struct timeval t1, t2;
    ha = new int[N];
    hb = new int[N];
    hc = new int[N];

    for (i = 0; i < N; i++)
    {
      ha[i] = rand () % 10000;
      hb[i] = rand () % 10000;
    }  
    gettimeofday(&t1, 0);
    for (i = 0; i < N; i++)
    {
      hc[i] = ha[i] + hb[i];
    }
    gettimeofday(&t2, 0);
    double time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0;

    printf("Se tardo :  %e ms \n", time);
}
